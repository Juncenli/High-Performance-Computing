#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

#include "math.h"
#include "time.h"
#include <iostream>
#include <fstream>
#include <iomanip>

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecaddgpu(float *d_a, float *d_b, float *d_c, int n) {
 //Device Code
 	//get unique index on which to compute
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	
	//add vectors on this index
	if(index < n){
		d_c[index] = d_a[index] + d_b[index];
	}
}

int main( int argc, char* argv[] ) {

    float timerValueGPU;
	hipEvent_t start, stop;
	hipEventCreate(&start); 
    hipEventCreate(&stop);

    // Size of vectors
    int n = 100000;
    float *h_a;
    float *h_b;
    float *h_c;
    float *d_a;
    float *d_b;
    float *d_c;
    // Allocate memory for each vector on host
    size_t bytes = n*sizeof(float);
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    // Initialize vectors on host
    int i;
    for ( i = 0; i < n; i++ ) {
        h_a[i] = sinf(i)*sinf(i);
        h_b[i] = cosf(i)*cosf(i);
    }
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
    // Execute the kernel
    int blockSize, gridSize;
    blockSize = 1024;
    gridSize = (int)ceil((float)n/blockSize);

    // start timer
	hipEventRecord(start, 0);
    // running kernel function
    vecaddgpu<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    // GPU computation time estimate
    hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueGPU, start, stop);
    printf("GPU calculation time %f msec\n", timerValueGPU);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
    // Sum up vector c and print result divided by n, this should equal 1 within error
    float sum = 0;
    for(i = 0; i < n; i++) {
        sum += h_c[i];
    }
    printf("final result: %f\n", sum/n);
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}