
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>


__global__ void computePi(float *pi, double n){
    //Global thread identifier
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    float i = 2 * thread_id + 1;
    int s = (1-2*(thread_id%2));
    float tmp;

    
    if(i <= (2 * n)){
        tmp = s * (4 / i);
        atomicAdd(pi, tmp);
    }

}

int main(int argc, char *argv[])
{
    //timing variables
    struct timeval  start, end;
    double comp_time;



    double n;// Number of iterations and control variable

    printf("Leibniz's series is used to approximate the number PI in a GPU implementation...\n");
    printf("Enter the number of iterations: ");    
    scanf("%lf",&n);
    printf("\nPlease wait. Running...\n");    

    //Start timer
    gettimeofday(&start, NULL);

    float* pi;
    hipMallocManaged(&pi, sizeof(float));
    *pi = 0.0;

    //Set dimensinos of blocks and grid
    int THREADS = 512;
    int BLOCKS = (n + THREADS - 1) / THREADS;

    computePi<<<BLOCKS, THREADS>>>(pi, n);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();

    //Stop timer
    gettimeofday(&end, NULL);
    comp_time = ((double) ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec))); 
    printf("\nAproximated value of PI = %1.16lf\n", *pi);
    printf("SINGLE precision total computing time for n=%.0lf: %f\n", n, comp_time);

    hipFree(pi);

    return 0;
}